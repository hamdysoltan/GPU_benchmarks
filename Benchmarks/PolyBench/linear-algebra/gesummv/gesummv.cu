#include "hip/hip_runtime.h"
/**
 * gesummv.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "gesummv.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 43532.0f
#define BETA 12313.0f

#define RUN_ON_CPU
FILE* fp ;

void gesummv(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NI,NI,n,n), DATA_TYPE POLYBENCH_2D(B,NI,NI,n,n), DATA_TYPE POLYBENCH_1D(tmp,NI,n),
		DATA_TYPE POLYBENCH_1D(x,NI,n), DATA_TYPE POLYBENCH_1D(y,NI,n))
{
	int i, j;
	
	for (i = 0; i < _PB_N; i++)
	{
		tmp[i] = 0;
		y[i] = 0;
		for (j = 0; j < _PB_N; j++)
		{
			tmp[i] = A[i][j] * x[j] + tmp[i];
			y[i] = B[i][j] * x[j] + y[i];
		}
		
		y[i] = alpha * tmp[i] + beta * y[i];
	}
}


void init(int n, DATA_TYPE *alpha, DATA_TYPE *beta, DATA_TYPE POLYBENCH_2D(A,NI,NI,n,n), DATA_TYPE POLYBENCH_2D(B,NI,NI,n,n), 
	DATA_TYPE POLYBENCH_1D(x,NI,n))
{
  	int i, j;

	*alpha = 43532;
	*beta = 12313;

 	for (i = 0; i < n; i++)
    	{
    		x[i] = ((DATA_TYPE) i) / NI;
      	
		for (j = 0; j < n; j++) 
		{
			A[i][j] = ((DATA_TYPE) i*j) / NI;
			B[i][j] = ((DATA_TYPE) i*j) / n;
		}
    }
}


void compareResults(int n, DATA_TYPE POLYBENCH_1D(y,NI,n), DATA_TYPE POLYBENCH_1D(y_outputFromGpu,NI,n))
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<n; i++) 
	{
		if (percentDiff(y[i], y_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gesummv_kernel(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* tmp, DATA_TYPE* x, DATA_TYPE* y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_N)
	{
		int j;
		for(j = 0; j < _PB_N; j++)
		{	
			tmp[i] += A[i * NI + j] * x[j];
			y[i] += B[i * NI + j] * x[j];
		}
		y[i] = alpha * tmp[i] + beta  * y[i];
	}
}

void gesummvCuda(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NI,NI,n,n), DATA_TYPE POLYBENCH_2D(B,NI,NI,n,n), 
		DATA_TYPE POLYBENCH_1D(tmp,NI,n), DATA_TYPE POLYBENCH_1D(x,NI,n), DATA_TYPE POLYBENCH_1D(y,NI,n),  
		DATA_TYPE POLYBENCH_1D(y_outputFromGpu,NI,n))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NI);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NI);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NI);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NI);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NI);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NI, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NI, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NI, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NI, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NI, hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)NI) / ((float)block.x) ), 1);


	/* Start timer. */
  	polybench_start_instruments;

	gesummv_kernel<<< grid, block>>>(n, alpha, beta, A_gpu, B_gpu, tmp_gpu, x_gpu, y_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NI, hipMemcpyDeviceToHost);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
// static
// void print_array(int n,
// 		 DATA_TYPE POLYBENICH_1D(y,NI,n))

// {
//   int i;

//   for (i = 0; i < n; i++) {
//     fprintf (stderr, DATA_PRINITF_MODIFIER, y[i]);
//     if (i % 20 == 0) fprintf (stderr, "\n");
//   }
// }


int main(int argc, char *argv[])
{
	/* Retrieve problem size. */
	int n = NI;
	fp = fopen("a.csv", "a");
     if (!fp) {
        // Error in file opening
        printf("Can't open file\n");
        
    }
	fprintf(fp, "%d\n", NI);
	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NI,n,n);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NI,NI,n,n);
	POLYBENCH_1D_ARRAY_DECL(tmp,DATA_TYPE,NI,n);
	POLYBENCH_1D_ARRAY_DECL(x,DATA_TYPE,NI,n);
	POLYBENCH_1D_ARRAY_DECL(y,DATA_TYPE,NI,n);
	POLYBENCH_1D_ARRAY_DECL(y_outputFromGpu,DATA_TYPE,NI,n);

	init(n, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(x));
	
	GPU_argv_init();
	gesummvCuda(n, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y),  
		POLYBENCH_ARRAY(y_outputFromGpu));
	
	// #ifdef RUN_ON_CPU

	// 	/* Start timer. */
	//   	polybench_start_instruments;

	// 	gesummv(n, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y));
		
	// 	/* Stop and print timer. */
	// 	printf("CPU Time in seconds:\n");
	//   	polybench_stop_instruments;
	//  	polybench_print_instruments;
	
	// 	compareResults(n, POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(y_outputFromGpu));

	// #else //prevent dead code elimination

	// 	polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(y_outputFromGpu)));

	// #endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);  
	POLYBENCH_FREE_ARRAY(tmp);
	POLYBENCH_FREE_ARRAY(x);  
	POLYBENCH_FREE_ARRAY(y);
	POLYBENCH_FREE_ARRAY(y_outputFromGpu);

	return 0;
}

#include <polybench.c>