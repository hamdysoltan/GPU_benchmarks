#include "hip/hip_runtime.h"
/**
 * mvt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "mvt.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU


void init_array(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n), DATA_TYPE POLYBENCH_1D(x1, N, n), DATA_TYPE POLYBENCH_1D(x2, N, n), DATA_TYPE POLYBENCH_1D(y1, N, n), DATA_TYPE POLYBENCH_1D(y2, N, n))
{
	int i, j;

	for (i = 0; i < n; i++)
	{
		x1[i] = ((DATA_TYPE) i) / N;
		x2[i] = ((DATA_TYPE) i + 1) / N;
		y1[i] = ((DATA_TYPE) i + 3) / N;
		y2[i] = ((DATA_TYPE) i + 4) / N;
		for (j = 0; j < n; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / N;
		}
	}
}



void runMvt(int n, DATA_TYPE POLYBENCH_2D(a, N, N, n, n), DATA_TYPE POLYBENCH_1D(x1, N, n), DATA_TYPE POLYBENCH_1D(x2, N, n), DATA_TYPE POLYBENCH_1D(y1, N, n), DATA_TYPE POLYBENCH_1D(y2, N, n))
{
	int i, j;
	
	for (i=0; i<_PB_N; i++) 
	{
		for (j=0; j<N; j++) 
		{
       		x1[i] = x1[i] + a[i][j] * y1[j];
        	}
    	}
	
	for (i=0; i<_PB_N; i++) 
	{
		for (j=0; j<_PB_N; j++) 
		{
 		      	x2[i] = x2[i] + a[j][i] * y2[j];
      		}
    	}
}


void compareResults(int n, DATA_TYPE POLYBENCH_1D(x1, N, n), DATA_TYPE POLYBENCH_1D(x1_outputFromGpu, N, n), DATA_TYPE POLYBENCH_1D(x2, N, n), DATA_TYPE POLYBENCH_1D(x2_outputFromGpu, N, n))
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<n; i++) 
	{
		if (percentDiff(x1[i], x1_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}

		if (percentDiff(x2[i], x2_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void mvt_kernel1(int n, DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_N)
	{
		int j;
		for(j=0; j < _PB_N; j++)
		{
			x1[i] += a[i * N + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2(int n, DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_N)
	{
		int j;
		for(j=0; j < _PB_N; j++)
		{
			x2[i] += a[j * N + i] * y_2[j];	
		}
	}
}

void mvtCuda(int n, DATA_TYPE POLYBENCH_2D(a, N, N, n, n), DATA_TYPE POLYBENCH_1D(x1, N, n), DATA_TYPE POLYBENCH_1D(x2, N, n), DATA_TYPE POLYBENCH_1D(y_1, N, n), DATA_TYPE POLYBENCH_1D(y_2, N, n), 
			DATA_TYPE POLYBENCH_1D(x1_outputFromGpu, N, n), DATA_TYPE POLYBENCH_1D(x2_outputFromGpu, N, n))
{
	DATA_TYPE* a_gpu;
	DATA_TYPE* x1_gpu;
	DATA_TYPE* x2_gpu;
	DATA_TYPE* y_1_gpu;
	DATA_TYPE* y_2_gpu;

	hipMalloc((void **)&a_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&x1_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&x2_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_1_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_2_gpu, sizeof(DATA_TYPE) * N);
	hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(x1_gpu, x1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(x2_gpu, x2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil((float)N/ ((float)DIM_THREAD_BLOCK_X)), 1);






	hipStream_t stream;
	hipStreamCreate(&stream);                                                                  // Create CUDA stream

	hipDeviceProp_t prop;                                                                        // CUDA device properties variable
	hipGetDeviceProperties( &prop, GPU_DEVICE);                                                 // Query GPU properties
	size_t size = min( int(prop.l2CacheSize * 0.75) , prop.persistingL2CacheMaxSize );
	hipDeviceSetLimit( cudaLimitPersistingL2CacheSize, size);                                  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed

	size_t window_size = min(prop.accessPolicyMaxWindowSize, 2097152); //2 MB                        // Select minimum of user defined num_bytes and max window size.

	hipLaunchAttributeValue stream_attribute1,stream_attribute2;                                                       // Stream level attributes data structure
	stream_attribute1.accessPolicyWindow.base_ptr  = reinterpret_cast<DATA_TYPE*>(a_gpu);               // Global Memory data pointer
	stream_attribute1.accessPolicyWindow.num_bytes = window_size;                                // Number of bytes for persistence access
	stream_attribute1.accessPolicyWindow.hitRatio  = 1;                                        // Hint for cache hit ratio
	stream_attribute1.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting;               // Persistence Property
	stream_attribute1.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;                // Type of access property on cache miss
    //hipLaunchAttributeID hipLaunchAttributeAccessPolicyWindow; 
	hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute1);   // Set the attributes to a CUDA Stream

	//for(int i = 0; i < 10; i++) {
	//	cuda_kernelA<<<grid_size,block_size,0,stream>>>(data1);                                 // This data1 is used by a kernel multiple times
	//}                                                                                           // [data1 + num_bytes) benefits from L2 persistence
	//cuda_kernelB<<<grid_size,block_size,0,stream>>>(data1);                                     // A different kernel in the same stream can also benefit
																								// from the persistence of data1
	//The following instrs are for disabling the presistence
	stream_attribute1.accessPolicyWindow.num_bytes = 0;                                          // Setting the window size to 0 disable it
	hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute1);   // Overwrite the access policy attribute to a CUDA Stream
	cudaCtxResetPersistingL2Cache();                                                            // Remove any persistent lines in L2 

	//cuda_kernelC<<<grid_size,block_size,0,stream>>>(data2);                                     // data2 can now benefit from full L2 in normal mode
    //----Ending  

	int gridd1=ceil((float)N/ ((float)DIM_THREAD_BLOCK_X));
	int blockx=DIM_THREAD_BLOCK_X;int blocky=DIM_THREAD_BLOCK_Y;
	printf("\n\n The blocks count is %d . The threads count_x is %d and the count_y is %d \n\n",gridd1,blockx,blocky);
	/* Start timer. */
  	polybench_start_instruments;
	
	mvt_kernel1<<<grid,block,0,stream>>>(n, a_gpu,x1_gpu,y_1_gpu);
	mvt_kernel2<<<grid,block,0,stream>>>(n, a_gpu,x2_gpu,y_2_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;
	
	hipMemcpy(x1_outputFromGpu, x1_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
	hipMemcpy(x2_outputFromGpu, x2_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);    
	
	hipFree(a_gpu);
	hipFree(x1_gpu);
	hipFree(x2_gpu);
	hipFree(y_1_gpu);
	hipFree(y_2_gpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
// static
// void print_array(int n,
// 		 DATA_TYPE POLYBENCH_1D(x1,N,n),
// 		 DATA_TYPE POLYBENCH_1D(x2,N,n))

// {
//   int i;

//   for (i = 0; i < n; i++) {
//     fprintf (stderr, DATA_PRINTF_MODIFIER, x1[i]);
//     fprintf (stderr, DATA_PRINTF_MODIFIER, x2[i]);
//     if (i % 20 == 0) fprintf (stderr, "\n");
//   }
// }


int main(int argc, char *argv[])
{
	int n = N;

	POLYBENCH_2D_ARRAY_DECL(a,DATA_TYPE,N,N,n,n);
	POLYBENCH_1D_ARRAY_DECL(x1,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(x2,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(x1_outputFromGpu,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(x2_outputFromGpu,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(y_1,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(y_2,DATA_TYPE,N,n);

	init_array(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x2), POLYBENCH_ARRAY(y_1), POLYBENCH_ARRAY(y_2));
	
	GPU_argv_init();

	mvtCuda(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x2), POLYBENCH_ARRAY(y_1), POLYBENCH_ARRAY(y_2), POLYBENCH_ARRAY(x1_outputFromGpu), POLYBENCH_ARRAY(x2_outputFromGpu));

	// #ifdef RUN_ON_CPU
	
	// 	/* Start timer. */
	//   	polybench_start_instruments;

	// 	//run the algorithm on the CPU
	// 	runMvt(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x2), POLYBENCH_ARRAY(y_1), POLYBENCH_ARRAY(y_2));

	// 	/* Stop and print timer. */
	// 	printf("CPU Time in seconds:\n");
	//   	polybench_stop_instruments;
	//  	polybench_print_instruments;
	
	// 	compareResults(n, POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x1_outputFromGpu), POLYBENCH_ARRAY(x2), POLYBENCH_ARRAY(x2_outputFromGpu));

	// #else //prevent dead code elimination

	// 	polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(x1_outputFromGpu), POLYBENCH_ARRAY(x2_outputFromGpu)));

	// #endif //RUN_ON_CPU

	POLYBENCH_FREE_ARRAY(a);
	POLYBENCH_FREE_ARRAY(x1);
	POLYBENCH_FREE_ARRAY(x2);
	POLYBENCH_FREE_ARRAY(x1_outputFromGpu);
	POLYBENCH_FREE_ARRAY(x2_outputFromGpu);
	POLYBENCH_FREE_ARRAY(y_1);
	POLYBENCH_FREE_ARRAY(y_2);

  	return 0;
}

#include <polybench.c>